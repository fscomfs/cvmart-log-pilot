#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <unistd.h>
#include <nvml.h>

int parseParameters(int argc, char *argv[],int &appNum,int &gpuInfo){
  int i = 0;
  if(argc == 1){
    return -1;
  }
  for (i = 1; i < argc; i++){
    if (strcmp(argv[i], "-appNum") == 0){
        appNum = (int)atoi(argv[++i]);
    }
    if (strcmp(argv[i], "-gpuInfo") == 0){
        gpuInfo = (int)atoi(argv[++i]);
    }
  }
  return 0;
}

__global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

int checkGpuAvailability(int appNum){
    int deviceCount = 0;
    //0 成功 1内存溢出无法使用 2 进程冲突 3 检测异常 4 没有GPU 5 INIT_ERROR
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
      printf("result-hipGetDeviceCount returned %d\n-> %s\n",
            static_cast<int>(error_id), hipGetErrorString(error_id));
      std::cout <<"result-Init error "<<"初始化失败"<<std::endl;
      exit(5);
    }
    if (deviceCount==0)
    {  
       std::cout <<"result-hipGetDeviceCount error "<<"没有检测到GPU"<<std::endl;
       exit(4);
    }
    
    int dev=0;
    int freeDeviceNum = 0;
    for (dev = 0; dev < deviceCount; ++dev) {
      hipSetDevice(dev);
      size_t avail;
	    size_t total;
      hipMemGetInfo(&avail, &total); 
      std::cout << "result-GPU-memory index-" <<dev<<":"<<(total-avail)/1024/1024<<"/"<<total/1024/1024<< std::endl;
      if (total==0)
      {
         std::cout <<"result-hipMemGetInfo error index-"<<dev<<",无法获取设备内存信息"<<std::endl;
         exit(1);
      }
      if (total>0&&(total-avail)/total>0.1)
      { 
        //如果过使用量大于 0.1 个百分比 就认为当前这个显卡是有人在使用
      }else{
        freeDeviceNum++;
      }
    }
    if (freeDeviceNum<appNum)
    {
      std::cout << "result-CheckGpu error 存在进程冲突可能: " <<appNum<<"/"<< freeDeviceNum << std::endl;
      exit(2);
    }
    
    dev=0;
    for (dev = 0; dev < deviceCount; ++dev){
      std::cout << "cudaSetDevice: " <<dev<< std::endl;
      hipError_t error_d = hipSetDevice(dev);
      if (error_d != hipSuccess) {
          std::cout <<"result-hipSetDevice error index-"<<dev<<",errorInfo:"<<hipGetErrorString(error_d)<<std::endl;
          exit(EXIT_FAILURE);
      }
      size_t avail;
	    size_t total;
      hipMemGetInfo(&avail, &total); 
      if (total==0)
      {
         std::cout <<"result-hipMemGetInfo error index-"<<dev<<",无法获取设备内存信息"<<std::endl;
         exit(EXIT_FAILURE);
      }
      if ((total-avail)/total>0.1)
      { 
          continue;
      }
      long N = 1 << 20;
      long nBytes = N * sizeof(float);
      // 申请host内存
      float *x, *y, *z;
      x = (float*)malloc(nBytes);
      y = (float*)malloc(nBytes);
      z = (float*)malloc(nBytes);

      // 初始化数据
      for (long i = 0; i < N; ++i)
      {
          x[i] = 10.0;
          y[i] = 20.0;
      }
      // 申请device内存
      float *d_x, *d_y, *d_z;
      //申请1G 的内存
      hipError_t error_x  = hipMalloc((void**)&d_x, nBytes);
      std::cout << "申请内存: " <<nBytes<< std::endl;
      if (error_x != hipSuccess) {
          std::cout <<"result-hipMalloc error index-"<<dev<<",errorInfo:"<<hipGetErrorString(error_x)<<std::endl;
          exit(EXIT_FAILURE);
      }
      //申请1G 的内存
      hipError_t error_y  = hipMalloc((void**)&d_y, nBytes);
      if (error_y != hipSuccess) {
          std::cout <<"result-hipMalloc error index-"<<dev<<",errorInfo:"<<hipGetErrorString(error_y)<<std::endl;
          exit(EXIT_FAILURE);
      }
      //申请1G 的内存
      hipError_t error_z  = hipMalloc((void**)&d_z, nBytes);
      if (error_z != hipSuccess) {
          std::cout <<"result-hipMalloc error index-"<<dev<<",errorInfo:"<<hipGetErrorString(error_z)<<std::endl;
          exit(EXIT_FAILURE);
      }
      // 释放device内存
      hipFree(d_x);
      hipFree(d_y);
      hipFree(d_z);
      // 释放host内存
      free(x);
      free(y);
      free(z);
    }
    return 0;
}


// int gpuInfo(int index){
//     // size_t avail;
// 	  // size_t total;
//     // cudaSetDevice(index);
//     // cudaMemGetInfo(&avail, &total); 
//     // std::cout <<"result-GPU-memory:"<<(total-avail)/1024/1024<<"/"<<total/1024/1024<<std::endl;
//     // sleep(5);
//     // return 0;

//    // nvmlInitWithFlags();
// }

int main(int argc, char *argv[])
{
    int appNum=0;
    int gpuInfoFlag = -1;
    if (-1==parseParameters(argc,argv,appNum,gpuInfoFlag))
    {
       printf("parameter_list failed!\n");
       exit(2);
    }
    printf("appNum:%d\n",appNum);
    printf("gpuInfoFlag:%d\n",gpuInfoFlag);
    return checkGpuAvailability(appNum);
}

